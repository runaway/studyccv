#include "hip/hip_runtime.h"
#undef USE_DISPATCH // nvcc doesn't support libdispatch
extern "C" {
#include "ccv.h"
}
#include <ctype.h>
#define CASE_TESTS // so that we don't include public available methods
#include "../lib/cuda/cwc_convnet.cu"
#include "../lib/ccv_convnet.c"

static const int DEVICE_COUNT = 4;

extern "C" void cwc_forwards_runtime(ccv_convnet_t* convnet, ccv_array_t* categorizeds, ccv_convnet_train_param_t params)
{
	int dual_batch = params.mini_batch;
	int mini_batch = dual_batch / DEVICE_COUNT;
	params.device_count = DEVICE_COUNT;
	_cwc_convnet_alloc_reserved_both(convnet, mini_batch, DEVICE_COUNT, params.layer_params);
	cwc_convnet_context_t* context = GPU(convnet)->contexts;
	int i, device_id;
	int conv_layers[] = {0, 3, 6, 7, 8};
	for (device_id = 0; device_id < DEVICE_COUNT; device_id++)
		for (i = 0; i < 5; i++)
		{
			ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
			EXTRA(layer)->vary.convolutional.forward.x = 4;
			EXTRA(layer)->vary.convolutional.forward.y = 8;
			EXTRA(layer)->vary.convolutional.forward.z = 32;
		}
	if (params.peer_access)
		_cwc_convnet_enable_peer_access(convnet, params.device_count);
	// doing model parallelism
	for (device_id = 0; device_id < DEVICE_COUNT; device_id++)
	{
		hipSetDevice(device_id);
		cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, 0, convnet->input, params.input.min_dim, params.input.max_dim, convnet->rows, convnet->cols, convnet->channels, 1000, 0, mini_batch, mini_batch * device_id, mini_batch, context->host[device_id].input, context->host[device_id].c);
		hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
		hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * mini_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
		assert(hipGetLastError() == hipSuccess);
	}
	for (device_id = 0; device_id < DEVICE_COUNT; device_id++)
	{
		hipSetDevice(device_id);
		hipDeviceSynchronize();
	}
	hipSetDevice(0);
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, context->device[0].data_stream);
	_cwc_convnet_encode_impl(convnet, DEVICE_COUNT, mini_batch, 0, context);
	for (device_id = 1; device_id < DEVICE_COUNT; device_id++)
	{
		hipSetDevice(device_id);
		hipEventRecord(context->device[device_id].data_joint, context->device[device_id].data_stream);
	}
	hipSetDevice(0);
	for (device_id = 1; device_id < DEVICE_COUNT; device_id++)
		hipStreamWaitEvent(context->device[0].data_stream, context->device[device_id].data_joint, 0);
	hipEventRecord(stop, context->device[0].data_stream);
	hipEventSynchronize(stop);
	float elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("%d GPUs uses %f ms\n", DEVICE_COUNT, elapsed_time);
	float *dual_out[DEVICE_COUNT] = {0};
	for (device_id = 0; device_id < DEVICE_COUNT; device_id++)
	{
		hipHostMalloc(&dual_out[device_id], sizeof(float) * dual_batch * 1000);
		hipSetDevice(device_id);
		hipMemcpy(dual_out[device_id], GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * 1000, hipMemcpyDeviceToHost);
	}
	ccv_convnet_compact(convnet);
	assert(hipGetLastError() == hipSuccess);
	// do it on one device
	device_id = 0;
	hipSetDevice(device_id);
	_cwc_convnet_alloc_reserved_both(convnet, dual_batch, 1, params.layer_params);
	assert(hipGetLastError() == hipSuccess);
	context = GPU(convnet)->contexts;
	for (i = 0; i < 5; i++)
	{
		ccv_convnet_layer_t* layer = GPU(convnet)->device[device_id].layers + conv_layers[i];
		EXTRA(layer)->vary.convolutional.forward.x = 4;
		EXTRA(layer)->vary.convolutional.forward.y = 8;
		EXTRA(layer)->vary.convolutional.forward.z = 32;
	}
	cwc_convnet_batch_formation(0, categorizeds, convnet->mean_activity, 0, 0, 0, 0, 0, convnet->input, params.input.min_dim, params.input.max_dim, convnet->rows, convnet->cols, convnet->channels, 1000, 0, dual_batch, 0, dual_batch, context->host[device_id].input, context->host[device_id].c);
	hipMemcpyAsync(context->device[device_id].input, context->host[device_id].input, sizeof(float) * convnet->rows * convnet->cols * convnet->channels * dual_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipMemcpyAsync(context->device[device_id].c, context->host[device_id].c, sizeof(int) * dual_batch, hipMemcpyHostToDevice, context->device[device_id].data_stream);
	assert(hipGetLastError() == hipSuccess);
	hipDeviceSynchronize();
	hipEventRecord(start, context->device[0].data_stream);
	_cwc_convnet_encode_impl(convnet, 1, dual_batch, 0, context);
	hipEventRecord(stop, context->device[0].data_stream);
	hipEventSynchronize(stop);
	elapsed_time = 0;
	hipEventElapsedTime(&elapsed_time, start, stop);
	printf("one GPU uses %f ms\n", elapsed_time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceSynchronize();
	float* out = 0;
	hipHostMalloc(&out, sizeof(float) * dual_batch * 1000);
	hipMemcpy(out, GPU(convnet)->device[device_id].forwards[convnet->count - 1], sizeof(float) * dual_batch * 1000, hipMemcpyDeviceToHost);
	ccv_convnet_free(convnet);
	int j;
	for (i = 0; i < 1000; i++)
	{
		for (device_id = 0; device_id < DEVICE_COUNT; device_id++)
				for (j = 0; j < mini_batch; j++)
					if (fabs(out[i * dual_batch + mini_batch * device_id + j] - dual_out[device_id][1000 * mini_batch * device_id + i * mini_batch + j]) > 1e-3)
						printf("%d %d %d %f %f\n", device_id, i, j, out[i * dual_batch + mini_batch * device_id + j], dual_out[device_id][1000 * mini_batch * device_id + i * mini_batch + j]);
	}
	for (device_id = 0; device_id < DEVICE_COUNT; device_id++)
		hipHostFree(dual_out[device_id]);
	hipHostFree(out);
}
